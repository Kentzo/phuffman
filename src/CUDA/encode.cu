#include "hip/hip_runtime.h"
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/for_each.h>
#include <thrust/remove.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <cmath>
#include <iostream>
#include "phuffman_math.cu"
#include "phuffman_limits.cuh"
#include "encode.hpp"
#include "../constants.h"
#include "../Code.h"

namespace phuffman {
    namespace CUDA {
        using thrust::device_ptr;
        using thrust::transform;
        using thrust::device_vector;
        using thrust::exclusive_scan;
        using thrust::for_each;
        using thrust::make_tuple;
        using thrust::get;
        using thrust::remove_if;
        using thrust::make_zip_iterator;
        using thrust::tuple;
        using std::cerr;
        using std::endl;
        typedef thrust::zip_iterator<tuple<device_vector<unsigned char>::iterator, device_vector<unsigned int>::iterator> > CharPosIterator;
        typedef thrust::tuple<unsigned char, unsigned int> CharPos;
        typedef thrust::device_vector<unsigned char> DevData;
        typedef thrust::device_vector<unsigned int> DevPrefixSum;

        __constant__ Code _CODES[ALPHABET_SIZE];

        struct _LengthEncode {
            /*!
              @abstract Returns number of bits for code of a given symbol.
              @discussion The result MUST lie within (0, MAXIMUM_CODELENGTH].
            */
            __device__ DevPrefixSum::value_type operator()(DevData::value_type symbol) {
                return _CODES[symbol].codelength;
            }
        };

        struct _NaiveMerge2 {
           unsigned int* _result;

            /*!
              @param result A pointer to the global memory. Memory MUST BE aligned by sizeof(uint2).
            */
            _NaiveMerge2(unsigned int* result) : _result(result) {}

            /*!
              @abstract Merges code of a given symbol into the device global memory at a given position.
              @param tuple A tuple that represents current symbol and position of its code at the device global memory.
            */
            __device__ void operator()(const CharPos& tuple) {
                Code code = _CODES[get<0>(tuple)];
                uint2 code_aligned = make_uint2(0, code.code) << (UINT2_BIT - code.codelength - (get<1>(tuple) % UINT_BIT));
                unsigned int* code_address = _result + get<1>(tuple) / UINT_BIT;
                atomicOr(code_address, code_aligned.x);
                atomicOr(code_address + 1, code_aligned.y);
            }
        };

        struct _IsConflictBlock {
            unsigned int _block_size_bit;

            _IsConflictBlock(unsigned int block_size) : _block_size_bit(bytes_to_bits(block_size)) {}
            __device__ bool operator()(const CharPos& tuple) {
                unsigned int start_block_idx = get<1>(tuple)/_block_size_bit;
                unsigned int code_address_end = get<1>(tuple) + _CODES[get<0>(tuple)].codelength;
                unsigned int end_block_idx = code_address_end / _block_size_bit;
                return start_block_idx != end_block_idx || (code_address_end % _block_size_bit) == 0;
            }
        };

        struct _CalcOffset {
            unsigned int _block_size_bit;

            _CalcOffset(unsigned int block_size) : _block_size_bit(bytes_to_bits(block_size)) {}
            __device__ unsigned char operator()(CharPos tuple) {
                return (get<1>(tuple) + _CODES[get<0>(tuple)].codelength) % _block_size_bit;
            }
        };

        void Encode(unsigned char* data, size_t length, CodesTable codes_table, unsigned int** result, size_t* result_length, size_t* result_length_bit,
                           unsigned int block_size /*= 0*/, unsigned char** block_offsets /*= NULL*/, size_t* block_offsets_length /*= NULL*/)
        {
            hipError_t error = hipSuccess;

            if ((error = hipMemcpyToSymbol(HIP_SYMBOL(_CODES), codes_table.codes, ALPHABET_SIZE * sizeof(Code))) != hipSuccess) {
                cerr << "Cannot copy codes into the device memory" << endl;
                throw error;
            }

            // Calculate Exclusive Prefix Sum
            DevData dev_data(data, data + length);
            DevPrefixSum dev_prefix_sum(length, 0);
            transform(dev_data.begin(), dev_data.end(), dev_prefix_sum.begin(), _LengthEncode());
            exclusive_scan(dev_prefix_sum.begin(), dev_prefix_sum.end(), dev_prefix_sum.begin());

            unsigned int* dev_result = NULL;
            try {
                // Encode Data
                // As we use exclusive prefix sum, we need to add the length of the last element manually
                size_t dev_result_length = dev_prefix_sum.back() + codes_table.codes[*(data + length - 1)].codelength; // bits
                dev_result_length = ceilf(static_cast<float>(dev_result_length) / UINT2_BIT) + 1; // uint2
                dev_result_length *= sizeof(uint2); // bytes

                if ((error = hipMalloc(&dev_result, dev_result_length)) != hipSuccess) {
                    cerr << "Cannot allocate " << dev_result_length << "bytes on the device" << endl;
                    throw error;
                }
                if ((error = hipMemset(dev_result, 0, dev_result_length)) != hipSuccess) {
                    cerr << "Cannot nullify " << dev_result_length << " bytes of memory at " << dev_result << endl;
                    throw error;
                }
                CharPosIterator charpos_begin(make_tuple(dev_data.begin(), dev_prefix_sum.begin())), charpos_end(make_tuple(dev_data.end(), dev_prefix_sum.end()));
                thrust::for_each(charpos_begin, charpos_end, _NaiveMerge2(dev_result));

                // Copy Data To Host
                *result = static_cast<unsigned int*>(calloc(dev_result_length, sizeof(unsigned char)));
                if ((error = hipMemcpy(*result, dev_result, dev_result_length, hipMemcpyDeviceToHost)) != hipSuccess) {
                    cerr << "Cannot copy data from device to host" << endl;
                    throw error;
                }

                // Get The Size Of The Result In Bits
                *result_length_bit = dev_prefix_sum.back() + codes_table.codes[*(data + length - 1)].codelength;
                *result_length = ceilf(static_cast<float>(*result_length_bit) / UINT_BIT);;

                // Calculate Block Offsets

                //            DevPrefixSum::iterator new_end = remove_if(dev_prefix_sum.begin(), dev_prefix_sum.end(), charpos_begin, _IsConflictBlock(block_size));
                //            *block_offsets = static_cast<unsigned char*>(calloc(new_end - dev_prefix_sum.begin(), sizeof(DevPrefixSum::value_type)));
                //            charpos_end = make_zip_iterator();

                //            transform(dev_prefix_sum.begin(), new_end, *block_offsets, _CalcOffset(block_size));
            }
            catch(...) {
                if (dev_result != NULL) {
                    hipFree(dev_result);
                }
                if (*result != NULL) {
                    free(*result);
                    *result = NULL;
                }
                *result_length_bit = 0;
                *result_length = 0;
                throw;
            }

            hipFree(dev_result);
        }
    }
}
